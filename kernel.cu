#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <>
#include <memory>

/*��CUDA�����л�ȡGPU�豸����*/

int main(void) {
	int device_count = 0;
	hipGetDeviceCount(&device_count);
	//�ú�������֧��CUDA��GPU�豸�ĸ���
	if (device_count ==0)
	{
		printf("There are no available device(s) that support CUDA\n");
	}
	else
	{
		printf("Detected %d CUDA Capable device(s)\n", device_count);
	}


	//ͨ���豸��Ϣ
	/*
	hipDeviceProp_t�ṹ���ṩ�˿�������ʶ���豸�Լ�ȷ��ʹ�õİ汾��Ϣ�����ԡ����ṩ��name���ԣ��������ַ���
	����ʽ�����豸�����ơ�������ͨ����ѯhipDriverGetVersion��hipRuntimeGetVersion���Ի���豸ʹ�õ�CUDA Driver
	������ʱ����İ汾������ж���豸����ϣ��ʹ�����еľ�����������������Ǹ��������ͨ��multiProcessorCount
	�������жϡ������Է����豸�ϵ����ദ����������������ͨ��ʹ��clockRate���Ի�ȡGPU��ʱ�����ʣ���KHz����ʱ��
	���ʡ�
	*/
	int device;
	hipDeviceProp_t device_Property;
	hipGetDevice(&device);
	hipGetDeviceProperties(&device_Property, device);
	printf("\nDevice %d:\"%s\"\n", device, device_Property.name);

	int driver_Version;
	int runtime_Version;
	hipDriverGetVersion(&driver_Version);
	hipRuntimeGetVersion(&runtime_Version);
	printf("CUDA Driver Version / Runtime Version %d.%d / %d.%d\n", driver_Version / 1000, (driver_Version % 100) / 10, runtime_Version / 1000, (runtime_Version % 100) / 10);
	printf("Total amount of global memory:%.0f Mbytes (%1lu bytes)\n", (float)device_Property.totalGlobalMem / 1048576.0f, (unsigned long long)device_Property.totalGlobalMem);
	printf("(%2d) Multiprocessors", device_Property.multiProcessorCount);
	printf("GPU Max Clock rate:%.0f MHz (%0.2f GHz)\n", device_Property.clockRate * 1e-3f, device_Property.clockRate * 1e-6f);


	/*
	����̶߳�����ʱ��ά�ģ�dim3���͡���ˣ����֪��ÿ��ά���п��Բ������������̺߳Ϳ顣����ÿ���ദ������
	�߳�������ÿ������߳�����Ҳ�����ơ�������ֿ���ͨ��maxThreadsPerMultiProcessor��maxThreadsPerBlock�ҵ���
	���ÿ�������������߳���������ÿ�����п��ܵ�����߳��������������ܱ�����
	����ͨ��maxThreadsDim��ȷ������ÿ��ά���ϵ�����߳�������ͬ����ÿ��ά����ÿ��������������ͨ��
	maxGridSize����ʶ�����Ƕ�����һ����������ֵ�����飬�ֱ���ʾx��y��zά���е����ֵ��
	*/

	printf("Maximum number of threads per multiprocessor:%d\n", device_Property.maxThreadsPerMultiProcessor);
	printf("Maximum number of threads per block:%d\n", device_Property.maxThreadsPerBlock);
	printf("Max dimension size of a thread block (x,y,z):(%d,%d,%d)\n", device_Property.maxThreadsDim[0], 
		device_Property.maxThreadsDim[1],
		device_Property.maxThreadsDim[2]);
	printf("Max dimension size of a grid size (x,y,z):(%d,%d,%d)\n", device_Property.maxGridSize[0],
		device_Property.maxGridSize[1],
		device_Property.maxGridSize[2]);
}